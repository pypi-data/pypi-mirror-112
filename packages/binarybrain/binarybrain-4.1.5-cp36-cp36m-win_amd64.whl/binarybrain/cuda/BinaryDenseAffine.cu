#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <algorithm>

#include "hip/hip_runtime.h"
#include ""

#include "bbcu/bbcu.h"
#include "bbcu/bbcu_util.h"

#include "Common.cuh"





// bit packing binary
template<typename T=float, int MAX_FRAME_UNIT=32, int MAX_NODE_UNIT=32>
__global__ void kernal_bit_BinaryDenseAffine_Forward
        (
            int const   *x_buf,
            int         *y_buf,
            T   const   *W_buf,
            T   const   *b_buf,
            T           *mean_buf,
            T           *rstd_buf,
            T           *running_mean_buf,
            T           *running_var_buf,
            T           gamma,
            T           beta,
            T           momentum,
            T           unbinarize_bias,
            T           reciprocal_frame_size,
            int         x_node_size,
            int         y_node_size,
            int         frame_size,
            int         frame_stride,
            int         lut_binarize
        )
{
    int y_node_id = threadIdx.y;
    int y_node    = blockIdx.y * blockDim.y + threadIdx.y;
    int id        = threadIdx.x;
    int id_step   = blockDim.x;

    __shared__  T           W[MAX_W_SIZE][MAX_NODE_UNIT];
                int   const *x_ptr[N];
                int         *y_ptr;
    
    if ( y_node < y_node_size ) {
        for (int x_node = id; x_node < x_node_size; x_node += id_step) {
            W[x_node][y_node_id] = W_buf[y_node*x_node_size + x_node]
        }
        b = b_buf[y_node];
    }

    __syncthreads();

    int bit_mask = (1 << (id & 0x1f));
    if ( y_node < y_node_size ) {
        for (int frame = id; frame < frame_size; frame += id_step) {
            T   y = b;
            for (int x_node = 0; x_node < x_node_size; ++x_node) {
                T   x = x_buf[frame_stride * x_node + (frame/32)];
                T   w = W[x_node][y_node_id];
                if ( x & bit_mask ) {
                    y += w;
                }
//              else {
//                  y -= w;
//              }
            }


        }
    }

    __syncthreads();



    if ( node < node_size ) {
        // read W
        for ( int i = id; i < (1 << N); i += id_step ) {
            W[i][node_id] = W_buf[node * (1 << N) + i];
            if ( lut_binarize ) {
                W[i][node_id] = W[i][node_id] > (T)0.5 ? (T)1.0 : (T)0.0;
            }
        }
        
        // read input index
        for ( int i = 0; i < N; ++i ) {
            x_ptr[i] = &x_buf[frame_stride * input_index[N*node + i]];
        }
                     
        y_ptr = &y_buf[node * frame_stride];
    }

    
    // 平均と分散計測
    T s1 = 0, c1 = 0, y1, t1;
    T s2 = 0, c2 = 0, y2, t2;
    for (int frame = id; frame < frame_size; frame += id_step) {
        if ( node < node_size ) {
            // Forward計算
            int bit  = (1 << (frame & 0x1f));
            int unit = (frame >> 5);
            T x[N];
            for ( int i = 0; i < N; ++i) {
                x[i] = (T)0.5 + ((x_ptr[i][unit] & bit) ? +unbinarize_bias : -unbinarize_bias);
            }
            T y = StochasticLut<N, T, MAX_NODE_UNIT>::NodeForward(node_id, x, W);
//          printf("[0] n=%3d f=%3d y=%10f\n", node, frame, y);

            // 集計
            y1 = y - c1;
            t1 = s1 + y1;
            c1 = (t1 - s1) - y1;
            s1 = t1;

            y2 = (y * y) - c2;
            t2 = s2 + y2;
            c2 = (t2 - s2) - y2;
            s2 = t2;
        }
    }

    s1 = device_LocalSumX<float>(s1, sbuf[node_id]);
    s2 = device_LocalSumX<float>(s2, sbuf[node_id]);

    T mean = s1 * reciprocal_frame_size;
    T var = max(1.0e-5f, (s2 * reciprocal_frame_size) - (mean * mean));
    T rstd = rsqrt(var);

//  if ( node < node_size && id == 0 ) {
////      printf("[0] n=%3d s1=%10f s2=%10f mean=%10f var=%10f rstd=%10f\n", node, s1, s2, mean, var, rstd);
//      printf("0\t%3d\t%.20e\t%.20e\t%.20e\t%.20e\t%.20e\n", node, s1, s2, mean, var, rstd);
//  }

    // 書き込み
    if (id == 0) {
        if ( node < node_size ) {
            running_mean_buf[node] = running_mean_buf[node] * momentum + mean * ((T)1.0 - momentum);
            running_var_buf[node]  = running_var_buf[node] * momentum + var * ((T)1.0 - momentum);
            mean_buf[node] = mean;
            rstd_buf[node] = rstd;
        }
    }

    // 正規化
    int loop_size = ((frame_size + blockDim.x - 1) & ~(blockDim.x - 1));
    for ( int frame = id; frame < loop_size; frame += id_step) {
        int unit     = (frame >> 5);
        int bit      = (frame & 0x1f);
        int bit_mask = (1 << bit);

        int y_mask = 0;
        if ( node < node_size && frame < frame_size) {
            // Forward計算
            T x[N];
            for ( int i = 0; i < N; ++i) {
                x[i] = (T)0.5 + ((x_ptr[i][unit] & bit_mask)  ? +unbinarize_bias : -unbinarize_bias);
            }
            T y = StochasticLut<N, T, MAX_NODE_UNIT>::NodeForward(node_id, x, W);

            y = (y - mean) * rstd;
            y = y * gamma + beta;

            if ( y > (T)0.5 ) {
                y_mask = bit_mask;
            }
        }

        y_mask = device_int_ShuffleOr(y_mask);

        if ( bit == 0 ) {
            if ( node < node_size && frame < frame_size ) {
                y_ptr[unit] = y_mask;
            }
        }
    }
}


BBCU_DLL_EXPORT int bbcu_bit_BinaryDenseAffine_Forward
        (
            int   const     *dev_x_buf,
            int             *dev_y_buf,
            float const     *dev_W,
            float           *dev_mean_buf,
            float           *dev_rstd_buf,
            float           *dev_running_mean_buf,
            float           *dev_running_var_buf,
            float           gamma,
            float           beta,
            float           momentum,
            float           unbinarize_bias,
            int             node_size,
            int             frame_size,
            int             frame_stride,
            int             lut_binarize,
            hipStream_t    streamId
        )
{
    BBCU_DEBUG_ASSERT(bbcu_IsDeviceAvailable());

    unsigned int const THREAD_SIZE    = 256;
    unsigned int const MAX_FRAME_UNIT = 256;
    unsigned int const MAX_NODE_UNIT  = 8;  // THREAD_SIZE/32 より小さくすること

#if 0
    dim3    block(MAX_FRAME_UNIT, THREAD_SIZE / MAX_FRAME_UNIT);
    while ( (int)block.x / 2 >= frame_size && block.x > 32 ) { block.x /= 2; block.y *= 2; }
    while ( (int)block.y / 2 >= node_size                  ) { block.y /= 2; }
#else
    dim3    block(THREAD_SIZE / MAX_NODE_UNIT, MAX_NODE_UNIT);
    while ( (int)block.y / 2 >= node_size  )                { block.y /= 2; block.x *= 2;}
    while ( (int)block.x / 2 >= frame_size && block.x > 32) { block.x /= 2; }
#endif

    block.x = std::min(block.x, MAX_FRAME_UNIT);
    block.y = std::min(block.y, MAX_NODE_UNIT);
    dim3    grid(1, (node_size + (block.y - 1)) / block.y);
    
    kernal_bit_DifferentiableLut_ForwardTraining<N, float, MAX_FRAME_UNIT, MAX_NODE_UNIT><<<grid, block, 0, streamId>>>(
            dev_x_buf,
            dev_y_buf,
            dev_input_index,
            dev_W,
            dev_mean_buf,
            dev_rstd_buf,
            dev_running_mean_buf,
            dev_running_var_buf,
            gamma,
            beta,
            momentum,
            unbinarize_bias,
            1.0f / (float)frame_size,
            node_size,
            frame_size,
            frame_stride,
            lut_binarize
        );
    BB_CUDA_CHECK_LAST_ERROR();
    
    return 0;
}




// end of file
